#include "hip/hip_runtime.h"
﻿/*
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include <stdio.h>
#include <math.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"

#endif


#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_impl.cuh"

#include ""
*/

#include <hip/hip_cooperative_groups.h>
#include <cstdlib>
#include <cstdio>
#include <math.h>
#include <string.h>

#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "defines.h"
#include "tables.h"

namespace cg = cooperative_groups;
#include "hip/hip_vector_types.h"
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"


hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__constant__ SimParams params;


struct integrate_functor
{
    float deltaTime;

    __host__ __device__
        integrate_functor(float delta_time) : deltaTime(delta_time) {}

    template <typename Tuple>
    __device__
        void operator()(Tuple t)
    {
        volatile float4 posData = thrust::get<0>(t);
        volatile float4 velData = thrust::get<1>(t);
        float3 pos = make_float3(posData.x, posData.y, posData.z);
        float3 vel = make_float3(velData.x, velData.y, velData.z);

        // vel += params.gravity * deltaTime;
        // vel *= params.globalDamping;

        // new position = old position + velocity * deltaTime
        pos += vel * deltaTime;
        //printf("pos=(%f, %f, %f)\n\n", pos.x, pos.y, pos.z);

        // set this to zero to disable collisions with cube sides
#if 1

        if (pos.x > 1.0f - params.particleRadius)
        {
            pos.x = 1.0f - params.particleRadius;
            vel.x *= params.boundaryDamping;
        }

        if (pos.x < -1.0f + params.particleRadius)
        {
            pos.x = -1.0f + params.particleRadius;
            vel.x *= params.boundaryDamping;
        }

        if (pos.y > 1.0f - params.particleRadius)
        {
            pos.y = 1.0f - params.particleRadius;
            vel.y *= params.boundaryDamping;
        }

        if (pos.z > 1.0f - params.particleRadius)
        {
            pos.z = 1.0f - params.particleRadius;
            vel.z *= params.boundaryDamping;
        }

        if (pos.z < -1.0f + params.particleRadius)
        {
            pos.z = -1.0f + params.particleRadius;
            vel.z *= params.boundaryDamping;
        }

#endif

        if (pos.y < -1.0f + params.particleRadius)
        {
            pos.y = -1.0f + params.particleRadius;
            vel.y *= params.boundaryDamping;
        }

        // store new position and velocity
        thrust::get<0>(t) = make_float4(pos, posData.w);
        thrust::get<1>(t) = make_float4(vel, velData.w);
    }
};

// calculate position in uniform grid
__device__ int3 calcGridPos(float3 p)
{
    int3 gridPos;
    gridPos.x = floorf((p.x - params.worldOrigin.x) / params.cellSize.x);
    gridPos.y = floorf((p.y - params.worldOrigin.y) / params.cellSize.y);
    gridPos.z = floorf((p.z - params.worldOrigin.z) / params.cellSize.z);
    return gridPos;
}

// calculate address in grid from position (clamping to edges)
__device__ uint calcGridHash(int3 gridPos)
{
    gridPos.x = gridPos.x & (params.gridSize.x - 1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (params.gridSize.y - 1);
    gridPos.z = gridPos.z & (params.gridSize.z - 1);
    return __umul24(__umul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __umul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

// calculate grid hash value for each particle
__global__
void calcHashD(uint* gridParticleHash,  // output
    uint* gridParticleIndex, // output
    float4* pos,               // input: positions
    uint    numParticles)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (index >= numParticles) return;

    volatile float4 p = pos[index];

    // get address in grid
    int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
    uint hash = calcGridHash(gridPos);

    // store grid hash and particle index
    gridParticleHash[index] = hash;
    gridParticleIndex[index] = index;
}

// rearrange particle data into sorted order, and find the start of each cell
// in the sorted hash array
__global__
void reorderDataAndFindCellStartD(uint* cellStart,        // output: cell start index
    uint* cellEnd,          // output: cell end index
    float4* sortedPos,        // output: sorted positions
    uint* gridParticleHash, // input: sorted grid hashes
    uint* gridParticleIndex,// input: sorted particle indices
    float4* oldPos,           // input: sorted position array
    uint    numParticles)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ uint sharedHash[];    // blockSize + 1 elements
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    uint hash;

    // handle case when no. of particles not multiple of block size
    if (index < numParticles)
    {
        hash = gridParticleHash[index];

        // Load hash data into shared memory so that we can look
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x + 1] = hash;

        if (index > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridParticleHash[index - 1];
        }
    }

    cg::sync(cta);

    if (index < numParticles)
    {
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell

        if (index == 0 || hash != sharedHash[threadIdx.x])
        {
            cellStart[hash] = index;

            if (index > 0)
                cellEnd[sharedHash[threadIdx.x]] = index;
        }

        if (index == numParticles - 1)
        {
            cellEnd[hash] = index + 1;
        }

        // Now use the sorted index to reorder the pos and vel data
        uint sortedIndex = gridParticleIndex[index];
        float4 pos = oldPos[sortedIndex];

        sortedPos[index] = pos;
    }
}


// Kernel function
__device__
float kernelPoly6(float3 rVector, float smoothingLength = 0.0476f)
{
    float poly6 = (315.f / (64.f * HIP_PI_F * pow(smoothingLength, 9.f))) * pow(pow(smoothingLength, 2.f) - pow(length(rVector), 2.f), 3.f);
    return poly6;   // for 0 < length(rVector) < supportRadius
}
__device__
float3 kernelPoly6Gradient(float3 rVector, float smoothingLength = 0.0476f)
{
    float3 poly6 = -(945.0f / (32.f * HIP_PI_F * pow(smoothingLength, 9.f))) * rVector * pow(pow(smoothingLength, 2.f) - dot(rVector, rVector), 2.f);
    return poly6;
}

__device__
float3 kernelSpikyGradient(float3 rVector, float smoothingLength = 0.0476f)
{
    if (length(rVector) != 0.0f)
    {
        float3 spiky = -(45.f / (HIP_PI_F * pow(smoothingLength, 6.f))) * pow(smoothingLength - length(rVector), 2.f) * normalize(rVector);
        return spiky;
    }
    else // ? 겹쳤을 때? particle이 사실 겹쳐진다는거 자체가 가능한 전제일까?
    {
        return make_float3(0.0f, 0.0f, 0.0f);
    } // 일단은 냅둬보고
}

__device__
float kernelViscosityLaplacian(float3 rVector, float smoothingLength = 0.0476)
{
    float viscosity = (45.f / (HIP_PI_F * pow(smoothingLength, 6.f))) * (smoothingLength - length(rVector));
    return viscosity;
}


// Compute Density and Pressure
__device__
float computeDensityByCell(int3    gridPos,
    uint    index,
    float3  indexPos,
    float3  indexVel,
    uint    originalIndex,
    float4* oldPosArray,
    float4* oldVelArray,
    uint* cellStart,
    uint* cellEnd,
    uint* gridParticleIndex,
    uint    numFluidParticles)
{
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = cellStart[gridHash];

    float density = 0.f;

    if (startIndex != 0xffffffff)          // cell is not empty
    {
        // iterate over particles in this cell
        uint endIndex = cellEnd[gridHash];

        for (uint j = startIndex; j < endIndex; j++)
        {
            float3 jPos = make_float3(oldPosArray[j]);
            // float3 jVel = make_float3(oldVelArray[j]);
            float3 iToj = indexPos - jPos;
            float length_iToj = length(iToj);

            if (length_iToj < params.smoothingLength)
            {
                if (originalIndex < numFluidParticles)
                    density += params.fluidParticleMass * kernelPoly6(iToj, params.smoothingLength);
                else
                    density += params.boundaryParticleMass * kernelPoly6(iToj, params.smoothingLength);
            }

            //if (j != index && length_iToj < params.smoothingLength)  // check not colliding with self and distance < h
            //{
            //    if (length_iToj < params.smoothingLength)
            //    {
            //        density += params.particleMass * kernelPoly6(iToj, params.smoothingLength);
            //    }
            //}
        }
    }

    return density;
}


__global__
void computeDensityAndPressureDevice(float* desities,
    float* pressures,
    float4* oldPosArray,               // input: sorted positions
    float4* oldVelArray,               // input: sorted velocities
    uint* gridParticleIndex,         // input: sorted particle indices
    uint* cellStart,
    uint* cellEnd,
    uint    numParticles,
    uint    numFluidParticles)
{
    uint index = (blockIdx.x * blockDim.x) + threadIdx.x; // It's me

    if (index >= numParticles) return;

    // read particle data from sorted arrays
    float3 indexPos = make_float3(oldPosArray[index]);
    float3 indexVel = make_float3(oldVelArray[index]);

    // get address in grid
    int3 gridPos = calcGridPos(indexPos);

    // examine neighbouring cells
    float density = 0.f;
    float pressure = 0.f;

    uint originalIndex = gridParticleIndex[index];

    for (int z = -1; z <= 1; z++)
    {
        for (int y = -1; y <= 1; y++)
        {
            for (int x = -1; x <= 1; x++)
            {
                int3 neighbourPos = gridPos + make_int3(x, y, z);
                density += computeDensityByCell(neighbourPos, index, indexPos, indexVel, originalIndex, oldPosArray, oldVelArray, cellStart, cellEnd, gridParticleIndex, numFluidParticles);
            }
        }
    }

    pressure = params.gasStiffnessConstant * (density - params.waterRestDensity);
    desities[originalIndex] = density;
    pressures[originalIndex] = pressure;
}


// Compute Froce
__device__
void computeForceAndViscosityByCell(float3* pressureForce,
    float3* viscosity,
    int3    gridPos,
    uint    index,
    float3  indexPos,
    float3  indexVel,
    uint    originalIndex,
    float4* oldPosArray,
    float4* oldVelArray,
    float* densities,
    float* pressures,
    uint* cellStart,
    uint* cellEnd,
    uint* gridParticleIndex,
    uint  numFluidParticles)
{
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = cellStart[gridHash];


    if (startIndex != 0xffffffff)          // cell is not empty
    {
        // iterate over particles in this cell
        uint endIndex = cellEnd[gridHash];

        for (uint j = startIndex; j < endIndex; j++)
        {
            uint jOriginalIndex = gridParticleIndex[j];
            uint originalIndex = gridParticleIndex[index];

            float3 jPos = make_float3(oldPosArray[j]);
            float3 jVel = make_float3(oldVelArray[j]);

            float3 iTojPos = indexPos - jPos;
            float lengthRij = length(iTojPos);
            float3 iTojVel = indexVel - jVel;
            // float lengthVij = length(iTojVel);

            if (j != index && lengthRij < params.smoothingLength)  // check not colliding with self and distance < h
            {
                if (originalIndex < numFluidParticles)
                { //fluid particle
                    *pressureForce += params.fluidParticleMass
                        * (pressures[originalIndex] / (densities[originalIndex] * densities[originalIndex]) + pressures[jOriginalIndex] / (densities[jOriginalIndex] * densities[jOriginalIndex]))
                        * kernelSpikyGradient(iTojPos, params.smoothingLength);

                    *viscosity += (params.fluidParticleMass / densities[originalIndex])
                        * (dot(iTojPos, iTojVel) / (dot(iTojPos, iTojPos) + 0.01f * params.smoothingLength * params.smoothingLength))
                        * kernelPoly6Gradient(iTojPos, params.smoothingLength);
                }
                else
                { //boundary particle
                    *pressureForce += params.boundaryParticleMass
                        * (pressures[originalIndex] / (densities[originalIndex] * densities[originalIndex]) + pressures[jOriginalIndex] / (densities[jOriginalIndex] * densities[jOriginalIndex]))
                        * kernelSpikyGradient(iTojPos, params.smoothingLength);

                    *viscosity += (params.boundaryParticleMass / densities[originalIndex])
                        * (dot(iTojPos, iTojVel) / (dot(iTojPos, iTojPos) + 0.01f * params.smoothingLength * params.smoothingLength))
                        * kernelPoly6Gradient(iTojPos, params.smoothingLength);
                }

            }
        }
    }
}

__global__
void computeForceAndViscosityDevice(float4* newVelocities,
    float4* entireForces,
    float   deltaTime,
    float* densities,
    float* pressures,
    float4* oldPosArray,               // input: sorted positions
    float4* oldVelArray,               // input: sorted velocities
    uint* gridParticleIndex,         // input: sorted particle indices
    uint* cellStart,
    uint* cellEnd,
    uint    numParticles,
    uint    numFluidParticles)
{
    uint index = (blockIdx.x * blockDim.x) + threadIdx.x; // It's me

    if (index >= numParticles) return;

    // read particle data from sorted arrays
    float3 indexPos = make_float3(oldPosArray[index]);
    float3 indexVel = make_float3(oldVelArray[index]);

    // get address in grid
    int3 gridPos = calcGridPos(indexPos);

    // examine neighbouring cells
    float3 pressureForce = make_float3(0.0f);
    float3 viscosity = make_float3(0.0f);
    float3 externalForce = make_float3(0.0f);
    float3 entireForce = make_float3(0.0f);

    uint originalIndex = gridParticleIndex[index];

    for (int z = -1; z <= 1; z++)
    {
        for (int y = -1; y <= 1; y++)
        {
            for (int x = -1; x <= 1; x++)
            {
                int3 neighbourPos = gridPos + make_int3(x, y, z);
                computeForceAndViscosityByCell(&pressureForce, &viscosity, neighbourPos, index, indexPos, indexVel, originalIndex, oldPosArray, oldVelArray, densities, pressures, cellStart, cellEnd, gridParticleIndex, numFluidParticles);
            }
        }
    }

    pressureForce *= -params.fluidParticleMass;
    viscosity *= params.fluidParticleMass * params.viscosityCoefficient * 10.f / densities[originalIndex]; // 2(d+2)
    externalForce = params.gravity * params.fluidParticleMass;  //params.gravity * densities[originalIndex]; // params.gravity * params.particleMass;
    entireForce = pressureForce + viscosity + externalForce;

    if (originalIndex < numFluidParticles)
    {
        entireForces[originalIndex] = entireForce;
        newVelocities[originalIndex] += make_float4(deltaTime * (entireForce / params.fluidParticleMass), 0.0f);
    }
    else
    {
        entireForces[originalIndex] = entireForce;
        newVelocities[originalIndex] += make_float4(deltaTime * (externalForce / params.BoundaryParticleMass), 0.0f);
    }
}

extern "C"
{
    void cudaInit(int argc, char** argv)
    {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        devID = findCudaDevice(argc, (const char**)argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void allocateArray(void** devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void* devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void threadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    void copyArrayToDevice(void* device, const void* host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char*)device + offset, host, size, hipMemcpyHostToDevice));
    }

    /*void registerGLBufferObject(uint vbo, struct hipGraphicsResource** cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
            cudaGraphicsMapFlagsNone));
    }*/

    void unregisterGLBufferObject(struct hipGraphicsResource* cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
    }

    void* mapGLBufferObject(struct hipGraphicsResource** cuda_vbo_resource)
    {
        void* ptr;
        checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
        size_t num_bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&ptr, &num_bytes,
            *cuda_vbo_resource));
        return ptr;
    }

    void unmapGLBufferObject(struct hipGraphicsResource* cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
    }

    void copyArrayFromDevice(void* host, const void* device,
        struct hipGraphicsResource** cuda_vbo_resource, int size)
    {
        if (cuda_vbo_resource)
        {
            device = mapGLBufferObject(cuda_vbo_resource);
        }

        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

        if (cuda_vbo_resource)
        {
            unmapGLBufferObject(*cuda_vbo_resource);
        }
    }

    void setParameters(SimParams* hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint& numBlocks, uint& numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    void integrateSystem(float* pos,
        float* vel,
        float deltaTime,
        uint numParticles)
    {
        thrust::device_ptr<float4> d_pos4((float4*)pos);
        thrust::device_ptr<float4> d_vel4((float4*)vel);

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4 + numParticles, d_vel4 + numParticles)),
            integrate_functor(deltaTime));
    }

    void calcHash(uint* gridParticleHash,
        uint* gridParticleIndex,
        float* pos,
        int    numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD << < numBlocks, numThreads >> > (gridParticleHash,
            gridParticleIndex,
            (float4*)pos,
            numParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    hipError_t reorderDataAndFindCellStart(uint* cellStart,
        uint* cellEnd,
        float* sortedPos,
        uint* gridParticleHash,
        uint* gridParticleIndex,
        float* oldPos,
        uint   numParticles,
        uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        hipError_t error = hipMemset(cellStart, 0xffffffff, numCells * sizeof(uint));
        checkCudaErrors(error);

        uint smemSize = sizeof(uint) * (numThreads + 1);
        reorderDataAndFindCellStartD << < numBlocks, numThreads, smemSize >> > (
            cellStart,
            cellEnd,
            (float4*)sortedPos,
            gridParticleHash,
            gridParticleIndex,
            (float4*)oldPos,
            numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");
        return error;
    }

    void computeDensityAndPressure(float* densities,
        float* pressures,
        float* sortedPos,
        float* sortedVel,
        uint* gridParticleIndex,
        uint* cellStart,
        uint* cellEnd,
        uint   numParticles,
        uint   numFluidParticles,
        uint   numCells)
    {
        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        computeDensityAndPressureDevice << < numBlocks, numThreads >> > (densities,
            pressures,
            (float4*)sortedPos,
            (float4*)sortedVel,
            gridParticleIndex,
            cellStart,
            cellEnd,
            numParticles,
            numFluidParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void computeForceAndViscosity(float* newVelocities,
        float* entireForce,
        float  deltaTime,
        float* desities,
        float* pressures,
        float* sortedPos,
        float* sortedVel,
        uint* gridParticleIndex,
        uint* cellStart,
        uint* cellEnd,
        uint   numParticles,
        uint   numFluidParticles,
        uint   numCells)
    {
        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        computeForceAndViscosityDevice << < numBlocks, numThreads >> > ((float4*)newVelocities,
            (float4*)entireForce,
            deltaTime,
            desities,
            pressures,
            (float4*)sortedPos,
            (float4*)sortedVel,
            gridParticleIndex,
            cellStart,
            cellEnd,
            numParticles,
            numFluidParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void sortParticles(uint* dGridParticleHash, uint* dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
            thrust::device_ptr<uint>(dGridParticleIndex));
    }


}   // extern "C"




// marching cubes
    // textures containing look-up tables
hipTextureObject_t gTriTex;
hipTextureObject_t gNumVerticesTex;

// volume data
hipTextureObject_t gVolumeTex;

// sample volume data set at a point
__device__
float CudaSampleVolume(hipTextureObject_t VolumeTex, uchar* Data, uint3 Position, uint3 GridSize)
{
    Position.x = min(Position.x, GridSize.x - 1);
    Position.y = min(Position.y, GridSize.y - 1);
    Position.z = min(Position.z, GridSize.z - 1);
    uint Index = (Position.z * GridSize.x * GridSize.y) + (Position.y * GridSize.x) + Position.x;
    //    return (float) data[i] / 255.0f;
    return tex1Dfetch<float>(VolumeTex, Index);
}

// compute position in 3d grid from 1d index
// only works for power of 2 sizes
__device__
uint3 CudaCalculateGridPosition(uint Index, uint3 GidSizeShift, uint3 GridSizeMask)
{
    uint3 GridPosition;
    GridPosition.x = Index & GridSizeMask.x;
    GridPosition.y = (Index >> GidSizeShift.y) & GridSizeMask.y;
    GridPosition.z = (Index >> GidSizeShift.z) & GridSizeMask.z;
    return GridPosition;
}

// evaluate Field function at point
__device__
float CudaParticleFieldFunction(float3 Vertex, float3 ParticlePosition)
{
    return lengthSquared(Vertex - ParticlePosition);
}

__device__
void CudaClassifyVoxelsByGrid(int3 GridPosition,
    float* OutField,
    float3 Position,
    float4* SortedPositions,
    float3 VoxelSize,
    uint* GridParticleIndices,
    uint* CellStarts,
    uint* CellEnds,
    uint NumFluidParticles,
    uint NumRenderingFluidParticles)
{
    uint GridHash = CudaCalculateGridHash(GridPosition);

    uint StartIndex = CellStarts[GridHash];

    if (StartIndex != 0xffffffff)
    {
        uint EndIndex = CellEnds[GridHash];

        for (uint NeighborIdx = StartIndex; NeighborIdx < EndIndex; ++NeighborIdx)
        {
            if (GridParticleIndices[NeighborIdx] >= NumRenderingFluidParticles)
            {
                continue;
            }
            float3 NeighborParticlePosition = make_float3(SortedPositions[NeighborIdx]);

            OutField[0] = min(CudaParticleFieldFunction(Position, NeighborParticlePosition), OutField[0]);
            OutField[1] = min(CudaParticleFieldFunction(Position + make_float3(VoxelSize.x, 0.0f, 0.0f), NeighborParticlePosition), OutField[1]);
            OutField[2] = min(CudaParticleFieldFunction(Position + make_float3(VoxelSize.x, VoxelSize.y, 0.0f), NeighborParticlePosition), OutField[2]);
            OutField[3] = min(CudaParticleFieldFunction(Position + make_float3(0.0f, VoxelSize.y, 0.0f), NeighborParticlePosition), OutField[3]);
            OutField[4] = min(CudaParticleFieldFunction(Position + make_float3(0.0f, 0.0f, VoxelSize.z), NeighborParticlePosition), OutField[4]);
            OutField[5] = min(CudaParticleFieldFunction(Position + make_float3(VoxelSize.x, 0.0f, VoxelSize.z), NeighborParticlePosition), OutField[5]);
            OutField[6] = min(CudaParticleFieldFunction(Position + make_float3(VoxelSize.x, VoxelSize.y, VoxelSize.z), NeighborParticlePosition), OutField[6]);
            OutField[7] = min(CudaParticleFieldFunction(Position + make_float3(0.0f, VoxelSize.y, VoxelSize.z), NeighborParticlePosition), OutField[7]);
            //if (gridPosition.x <= 7 && gridPosition.y <= 7 && gridPosition.z <= 7)
            //{
            //    printf("\Alpha[%u: %u - %u], neighbor(%u, %u, %u): Position=(%f, %f, %f) particle=(%f + %f, %f + %f, %f + %f) Field: %f, %f, %f, %f, %f, %f, %f, %f\OutNormalVector",
            //        Index, StartIndex, EndIndex,
            //        gridPosition.x, gridPosition.y, gridPosition.z,
            //        position.x, VoxelSize.x, 
            //        position.y, VoxelSize.y, 
            //        position.z, VoxelSize.z,
            //        ParticlePosition.x, ParticlePosition.y, ParticlePosition.z,
            //        outField[0],
            //        outField[1],
            //        outField[2],
            //        outField[3],
            //        outField[4],
            //        outField[5],
            //        outField[6],
            //        outField[7]
            //    );
            //}
        }
    }
}

// classify voxel based on number of vertices it will generate
// one thread per voxel
__global__
void CudaClassifyVoxels(uint* OutVoxelVertices,
    uint* OutOccupiedVoxels,
    uchar* Volumes,
    uint3               GridSize,
    uint3               GridSizeShift,
    uint3               GridSizeMask,
    uint                NumVoxels,
    float3              VoxelSize,
    float               IsoValue,
    hipTextureObject_t NumVerticesTex,
    hipTextureObject_t VolumeTex,
    float4* SortedPositions,
    uint* GridParticleIndices,
    uint* CellStarts,
    uint* CellEnds,
    uint                NumFluidParticles,
    uint                NumRenderingFluidParticles)
{
    uint BlockIdx = (blockIdx.y * gridDim.x) + blockIdx.x;
    uint Index = (BlockIdx * blockDim.x) + threadIdx.x;

    uint3 GridPosition = CudaCalculateGridPosition(Index, GridSizeShift, GridSizeMask);
    //if (GridPosition.x < 26 && GridPosition.y < 26 && GridPosition.z < 26)
    //{
    //    printf("[%u]: grid=(%u, %u, %u)\OutNormalVector", Index,
    //        GridPosition.x, GridPosition.y, GridPosition.z);
    //}
    // read Field values at neighbouring grid vertices

    float3 Position;
    Position.x = -1.0f + (GridPosition.x * VoxelSize.x);
    Position.y = -1.0f + (GridPosition.y * VoxelSize.y);
    Position.z = -1.0f + (GridPosition.z * VoxelSize.z);

#if SAMPLE_VOLUME
    float Field[8];
    Field[0] = CudaSampleVolume(VolumeTex, Volumes, GridPosition, GridSize);
    Field[1] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(1, 0, 0), GridSize);
    Field[2] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(1, 1, 0), GridSize);
    Field[3] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(0, 1, 0), GridSize);
    Field[4] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(0, 0, 1), GridSize);
    Field[5] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(1, 0, 1), GridSize);
    Field[6] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(1, 1, 1), GridSize);
    Field[7] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(0, 1, 1), GridSize);
#else
    float Field[8] = { FLT_MAX,
                       FLT_MAX,
                       FLT_MAX,
                       FLT_MAX,

                       FLT_MAX,
                       FLT_MAX,
                       FLT_MAX,
                       FLT_MAX, };

    for (int z = -gParameters.MarchingCubesNeighborSearchDepth; z <= gParameters.MarchingCubesNeighborSearchDepth; ++z)
    {
        for (int y = -gParameters.MarchingCubesNeighborSearchDepth; y <= gParameters.MarchingCubesNeighborSearchDepth; ++y)
        {
            for (int x = -gParameters.MarchingCubesNeighborSearchDepth; x <= gParameters.MarchingCubesNeighborSearchDepth; ++x)
            {
                int3 NeighborPosition = make_int3(GridPosition) + make_int3(x, y, z);

                CudaClassifyVoxelsByGrid(NeighborPosition,
                    Field,
                    Position,
                    SortedPositions,
                    VoxelSize,
                    GridParticleIndices,
                    CellStarts,
                    CellEnds,
                    NumFluidParticles,
                    NumRenderingFluidParticles);
            }
        }
    }
#endif

    // calculate flag indicating if each Vertex is inside or outside isosurface
    uint CubeIndex;
    CubeIndex = uint(Field[0] < IsoValue);
    CubeIndex += uint(Field[1] < IsoValue) * 2u;
    CubeIndex += uint(Field[2] < IsoValue) * 4u;
    CubeIndex += uint(Field[3] < IsoValue) * 8u;
    CubeIndex += uint(Field[4] < IsoValue) * 16u;
    CubeIndex += uint(Field[5] < IsoValue) * 32u;
    CubeIndex += uint(Field[6] < IsoValue) * 64u;
    CubeIndex += uint(Field[7] < IsoValue) * 128u;
    //if (GridPosition.x <= 7 && GridPosition.y <= 7 && GridPosition.z <= 7)
    //{
    //if (CubeIndex != 0u)
    //{
    //    printf("[%u]: grid(%u, %u, %u) cube index %u%u%u%u %u%u%u%u=%u\OutNormalVector", Index,
    //        GridPosition.x, GridPosition.y, GridPosition.z,
    //        !!(CubeIndex & 1u),
    //        !!(CubeIndex & 1u << 1u),
    //        !!(CubeIndex & 1u << 2u),
    //        !!(CubeIndex & 1u << 3u),
    //        !!(CubeIndex & 1u << 4u),
    //        !!(CubeIndex & 1u << 5u),
    //        !!(CubeIndex & 1u << 6u),
    //        !!(CubeIndex & 1u << 7u),
    //        CubeIndex
    //    );
    //}
    //}


    // read number of vertices from texture
    uint NumVertices = tex1Dfetch<uint>(NumVerticesTex, CubeIndex);

    if (Index < NumVoxels)
    {
        OutVoxelVertices[Index] = NumVertices;
        OutOccupiedVoxels[Index] = (NumVertices > 0);
    }
}

// compact voxel array
__global__
void CudaCompactVoxels(uint* OutCompactedVoxelArray, uint* OccupiedVoxels, uint* OccupiedScanVoxels, uint NumVoxels)
{
    uint BlockIdx = (blockIdx.y * gridDim.x) + blockIdx.x;
    uint Index = (BlockIdx * blockDim.x) + threadIdx.x;

    if (OccupiedVoxels[Index] && (Index < NumVoxels))
    {
        OutCompactedVoxelArray[OccupiedScanVoxels[Index]] = Index;
    }
}

// compute interpolated Vertex along an edge
__device__
float3 CudaVertexLerp(float Isolevel, float3 Position0, float3 Position1, float Alpha0, float Alpha1)
{
    float Alpha = (Isolevel - Alpha0) / (Alpha1 - Alpha0);
    return lerp(Position0, Position1, Alpha);
}

// compute interpolated Vertex position and normal along an edge
__device__
void CudaVertexLerp2(float Isolevel, float3 Position0, float3 Position1, float4 AlphaVector0, float4 AlphaVector1, float3& OutPosition, float3& OutNormalVector)
{
    float Alpha = (Isolevel - AlphaVector0.w) / (AlphaVector1.w - AlphaVector0.w);
    OutPosition = lerp(Position0, Position1, Alpha);
    OutPosition = make_float3(OutPosition.x * -1.0f, OutPosition.z * -1.0f, OutPosition.y * -1.0f);
    OutNormalVector.x = -lerp(AlphaVector0.x, AlphaVector1.x, Alpha);
    OutNormalVector.y = -lerp(AlphaVector0.z, AlphaVector1.z, Alpha);
    OutNormalVector.z = -lerp(AlphaVector0.y, AlphaVector1.y, Alpha);
    //printf("\Alpha\tinterpolate: Alpha=%f, OutNormalVector=(%f, %f, %f), Alpha0=(%f, %f, %f), Alpha1=(%f, %f, %f)\OutNormalVector", Alpha, 
    //    OutNormalVector.x, OutNormalVector.y, OutNormalVector.z, Alpha0.x, Alpha0.y, Alpha0.z, Alpha1.x, Alpha1.y, Alpha1.z);
    //    OutNormalVector = normalize(OutNormalVector);
}

__device__
float4 CudaParticleFieldFunction4(float3 Vertex, float3 ParticlePosition)
{
    float Value = CudaParticleFieldFunction(Vertex, ParticlePosition);
    const float Delta = 0.001f;
    float Dx = CudaParticleFieldFunction(make_float3(Vertex.x + Delta, Vertex.y, Vertex.z), ParticlePosition) - Value;
    float Dy = CudaParticleFieldFunction(make_float3(Vertex.x, Vertex.y + Delta, Vertex.z), ParticlePosition) - Value;
    float Dz = CudaParticleFieldFunction(make_float3(Vertex.x, Vertex.y, Vertex.z + Delta), ParticlePosition) - Value;
    //printf("\Alpha\tv: %f, dx dy dz = (%f, %f, %f)\OutNormalVector", Value, dx, dy, dz);
    return make_float4(Dx, Dy, Dz, Value);
}

// evaluate Field function at a point
// returns value and gradient in float4
__device__
void CudaGetNormalVector(float4& OutNormalVector, float3 Vertex, float3 ParticlePosition)
{
    float3 ParticleToVertex = Vertex - ParticlePosition;
    if (OutNormalVector.w > lengthSquared(ParticleToVertex) - gParameters.SupportRadiusSquared)
    {
        OutNormalVector = make_float4(normalize(ParticleToVertex), lengthSquared(ParticleToVertex) - gParameters.SupportRadiusSquared);
    }
}

__device__
void CudaGenerateTrianglesByGrid(int3 GridPosition,
    float4* OutField,
    float3* Verticess,
    float4* SortedPositions,
    float3 VoxelSize,
    uint* GridParticleIndices,
    uint* CellStarts,
    uint* CellEnds,
    uint NumFluidParticles,
    uint NumRenderingFluidParticles)
{
    uint GridHash = CudaCalculateGridHash(GridPosition);

    uint StartIndex = CellStarts[GridHash];

    if (StartIndex != 0xffffffff)
    {
        uint EndIndex = CellEnds[GridHash];
        for (uint NeighborIdx = StartIndex; NeighborIdx < EndIndex; ++NeighborIdx)
        {
            if (GridParticleIndices[NeighborIdx] >= NumRenderingFluidParticles)
            {
                continue;
            }
            float3 NeighborParticlePosition = make_float3(SortedPositions[NeighborIdx]);

            //getNormalVector(outField[j], vertices[j], ParticlePosition);
            float4 Gradient = CudaParticleFieldFunction4(Verticess[0], NeighborParticlePosition);
            if (OutField[0].w > Gradient.w)
            {
                OutField[0] = Gradient;
            }
            Gradient = CudaParticleFieldFunction4(Verticess[1], NeighborParticlePosition);
            if (OutField[1].w > Gradient.w)
            {
                OutField[1] = Gradient;
            }
            Gradient = CudaParticleFieldFunction4(Verticess[2], NeighborParticlePosition);
            if (OutField[2].w > Gradient.w)
            {
                OutField[2] = Gradient;
            }
            Gradient = CudaParticleFieldFunction4(Verticess[3], NeighborParticlePosition);
            if (OutField[3].w > Gradient.w)
            {
                OutField[3] = Gradient;
            }
            Gradient = CudaParticleFieldFunction4(Verticess[4], NeighborParticlePosition);
            if (OutField[4].w > Gradient.w)
            {
                OutField[4] = Gradient;
            }
            Gradient = CudaParticleFieldFunction4(Verticess[5], NeighborParticlePosition);
            if (OutField[5].w > Gradient.w)
            {
                OutField[5] = Gradient;
            }
            Gradient = CudaParticleFieldFunction4(Verticess[6], NeighborParticlePosition);
            if (OutField[6].w > Gradient.w)
            {
                OutField[6] = Gradient;
            }
            Gradient = CudaParticleFieldFunction4(Verticess[7], NeighborParticlePosition);
            if (OutField[7].w > Gradient.w)
            {
                OutField[7] = Gradient;
            }
        }
    }
}

// generate triangles for each voxel using marching cubes
// interpolates normals from Field function
__global__ void
CudaGenerateTriangles(float4* OutPositions,
    float4* OutNormals,
    uint* CompactedVoxelArray,
    uint* NumScannedVertices,
    uint3 GridSize,
    uint3 GridSizeShift,
    uint3 GridSizeMask,
    float3 VoxelSize,
    float IsoValue,
    uint NumActiveVoxels,
    uint NumMaxVertices,
    hipTextureObject_t TriTex,
    hipTextureObject_t NumVerticesTex,
    float4* SortedPositions,
    uint* GridParticleIndices,
    uint* CellStarts,
    uint* CellEnds,
    uint NumFluidParticles,
    uint NumRenderingFluidParticles)
{
    uint BlockIdx = (blockIdx.y * gridDim.x) + blockIdx.x;
    uint Index = (BlockIdx * blockDim.x) + threadIdx.x;

    if (Index > NumActiveVoxels - 1)
    {
        // can'Alpha return here because of syncthreads()
        Index = NumActiveVoxels - 1;
    }

#if SKIP_EMPTY_VOXELS
    uint Voxel = CompactedVoxelArray[Index];
#else
    uint Voxel = OutputIdx;
#endif

    // compute position in 3d grid
    uint3 GridPosition = CudaCalculateGridPosition(Voxel, GridSizeShift, GridSizeMask);

    float3 Position;
    Position.x = -1.0f + (GridPosition.x * VoxelSize.x);
    Position.y = -1.0f + (GridPosition.y * VoxelSize.y);
    Position.z = -1.0f + (GridPosition.z * VoxelSize.z);

    // calculate cell Vertex positions
    float3 Vertices[8];
    Vertices[0] = Position;
    Vertices[1] = Position + make_float3(VoxelSize.x, 0, 0);
    Vertices[2] = Position + make_float3(VoxelSize.x, VoxelSize.y, 0);
    Vertices[3] = Position + make_float3(0, VoxelSize.y, 0);
    Vertices[4] = Position + make_float3(0, 0, VoxelSize.z);
    Vertices[5] = Position + make_float3(VoxelSize.x, 0, VoxelSize.z);
    Vertices[6] = Position + make_float3(VoxelSize.x, VoxelSize.y, VoxelSize.z);
    Vertices[7] = Position + make_float3(0, VoxelSize.y, VoxelSize.z);

    // evaluate Field values
    float4 Field[8] = {
        make_float4(0.0f, 0.0f, 0.0f, FLT_MAX),
        make_float4(0.0f, 0.0f, 0.0f, FLT_MAX),
        make_float4(0.0f, 0.0f, 0.0f, FLT_MAX),
        make_float4(0.0f, 0.0f, 0.0f, FLT_MAX),

        make_float4(0.0f, 0.0f, 0.0f, FLT_MAX),
        make_float4(0.0f, 0.0f, 0.0f, FLT_MAX),
        make_float4(0.0f, 0.0f, 0.0f, FLT_MAX),
        make_float4(0.0f, 0.0f, 0.0f, FLT_MAX)
    };
    //Field[0] = fieldFunc4(Value[0]);
    //Field[1] = fieldFunc4(Value[1]);
    //Field[2] = fieldFunc4(Value[2]);
    //Field[3] = fieldFunc4(Value[3]);
    //Field[4] = fieldFunc4(Value[4]);
    //Field[5] = fieldFunc4(Value[5]);
    //Field[6] = fieldFunc4(Value[6]);
    //Field[7] = fieldFunc4(Value[7]);

    for (int z = -gParameters.MarchingCubesNeighborSearchDepth; z <= gParameters.MarchingCubesNeighborSearchDepth; ++z)
    {
        for (int y = -gParameters.MarchingCubesNeighborSearchDepth; y <= gParameters.MarchingCubesNeighborSearchDepth; ++y)
        {
            for (int x = -gParameters.MarchingCubesNeighborSearchDepth; x <= gParameters.MarchingCubesNeighborSearchDepth; ++x)
            {
                int3 NeighborPosition = make_int3(GridPosition) + make_int3(x, y, z);
                CudaGenerateTrianglesByGrid(NeighborPosition,
                    Field,
                    Vertices,
                    SortedPositions,
                    VoxelSize,
                    GridParticleIndices,
                    CellStarts,
                    CellEnds,
                    NumFluidParticles,
                    NumRenderingFluidParticles);
            }
        }
    }

    // recalculate flag
    // (this is faster than storing it in global memory)
    uint CubeIndex;
    CubeIndex = uint(Field[0].w < IsoValue);
    CubeIndex += uint(Field[1].w < IsoValue) * 2u;
    CubeIndex += uint(Field[2].w < IsoValue) * 4u;
    CubeIndex += uint(Field[3].w < IsoValue) * 8u;
    CubeIndex += uint(Field[4].w < IsoValue) * 16u;
    CubeIndex += uint(Field[5].w < IsoValue) * 32u;
    CubeIndex += uint(Field[6].w < IsoValue) * 64u;
    CubeIndex += uint(Field[7].w < IsoValue) * 128u;
    if (CubeIndex != 0u)
    {
        //printf("[%u]: triangle grid(%u, %u, %u) cube index %u%u%u%u %u%u%u%u=%u\OutNormalVector", Index,
        //    GridPosition.x, GridPosition.y, GridPosition.z,
        //    !!(CubeIndex & 1u),
        //    !!(CubeIndex & 1u << 1u),
        //    !!(CubeIndex & 1u << 2u),
        //    !!(CubeIndex & 1u << 3u),
        //    !!(CubeIndex & 1u << 4u),
        //    !!(CubeIndex & 1u << 5u),
        //    !!(CubeIndex & 1u << 6u),
        //    !!(CubeIndex & 1u << 7u),
        //    CubeIndex
        //);
    }


    // find the vertices where the surface intersects the cube

#if USE_SHARED
    // use partioned shared memory to avoid using local memory
    __shared__ float3 VerticesList[12 * NTHREADS];
    __shared__ float3 NormalsList[12 * NTHREADS];

    CudaVertexLerp2(IsoValue, Vertices[0], Vertices[1], Field[0], Field[1], VerticesList[threadIdx.x], NormalsList[threadIdx.x]);
    CudaVertexLerp2(IsoValue, Vertices[1], Vertices[2], Field[1], Field[2], VerticesList[threadIdx.x + NTHREADS], NormalsList[threadIdx.x + NTHREADS]);
    CudaVertexLerp2(IsoValue, Vertices[2], Vertices[3], Field[2], Field[3], VerticesList[threadIdx.x + (NTHREADS * 2)], NormalsList[threadIdx.x + (NTHREADS * 2)]);
    CudaVertexLerp2(IsoValue, Vertices[3], Vertices[0], Field[3], Field[0], VerticesList[threadIdx.x + (NTHREADS * 3)], NormalsList[threadIdx.x + (NTHREADS * 3)]);
    CudaVertexLerp2(IsoValue, Vertices[4], Vertices[5], Field[4], Field[5], VerticesList[threadIdx.x + (NTHREADS * 4)], NormalsList[threadIdx.x + (NTHREADS * 4)]);
    CudaVertexLerp2(IsoValue, Vertices[5], Vertices[6], Field[5], Field[6], VerticesList[threadIdx.x + (NTHREADS * 5)], NormalsList[threadIdx.x + (NTHREADS * 5)]);
    CudaVertexLerp2(IsoValue, Vertices[6], Vertices[7], Field[6], Field[7], VerticesList[threadIdx.x + (NTHREADS * 6)], NormalsList[threadIdx.x + (NTHREADS * 6)]);
    CudaVertexLerp2(IsoValue, Vertices[7], Vertices[4], Field[7], Field[4], VerticesList[threadIdx.x + (NTHREADS * 7)], NormalsList[threadIdx.x + (NTHREADS * 7)]);
    CudaVertexLerp2(IsoValue, Vertices[0], Vertices[4], Field[0], Field[4], VerticesList[threadIdx.x + (NTHREADS * 8)], NormalsList[threadIdx.x + (NTHREADS * 8)]);
    CudaVertexLerp2(IsoValue, Vertices[1], Vertices[5], Field[1], Field[5], VerticesList[threadIdx.x + (NTHREADS * 9)], NormalsList[threadIdx.x + (NTHREADS * 9)]);
    CudaVertexLerp2(IsoValue, Vertices[2], Vertices[6], Field[2], Field[6], VerticesList[threadIdx.x + (NTHREADS * 10)], NormalsList[threadIdx.x + (NTHREADS * 10)]);
    CudaVertexLerp2(IsoValue, Vertices[3], Vertices[7], Field[3], Field[7], VerticesList[threadIdx.x + (NTHREADS * 11)], NormalsList[threadIdx.x + (NTHREADS * 11)]);
    //for (uint Index = 0; Index < 12; ++Index)
    //{
    //    if (normlist[threadIdx.x + (NTHREADS * Index)].x == 0.0f && normlist[threadIdx.x + (NTHREADS * Index)].y == 0.0f && normlist[threadIdx.x + (NTHREADS * Index)].z == 0.0f)
    //    {
    //        printf("\Alpha[%u] norm[%u]: (%f, %f, %f)\OutNormalVector", Index, threadIdx.x + (NTHREADS * Index), 
    //            normlist[threadIdx.x + (NTHREADS * Index)].x, normlist[threadIdx.x + (NTHREADS * Index)].y, normlist[threadIdx.x + (NTHREADS * Index)].z);
    //    }
    //}
    //printf("\Alpha[%6u] norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector"
    //    "\Alpha         norm[%u]: (%f, %f, %f)\OutNormalVector", Index,
    //    threadIdx.x, normlist[threadIdx.x].x, normlist[threadIdx.x].y, normlist[threadIdx.x].z,
    //    threadIdx.x + NTHREADS, normlist[threadIdx.x + NTHREADS].x, normlist[threadIdx.x + NTHREADS].y, normlist[threadIdx.x + NTHREADS].z,
    //    threadIdx.x + (NTHREADS * 2), normlist[threadIdx.x + (NTHREADS * 2)].x, normlist[threadIdx.x + (NTHREADS * 2)].y, normlist[threadIdx.x + (NTHREADS * 2)].z,
    //    threadIdx.x + (NTHREADS * 3), normlist[threadIdx.x + (NTHREADS * 3)].x, normlist[threadIdx.x + (NTHREADS * 3)].y, normlist[threadIdx.x + (NTHREADS * 3)].z,
    //    threadIdx.x + (NTHREADS * 4), normlist[threadIdx.x + (NTHREADS * 4)].x, normlist[threadIdx.x + (NTHREADS * 4)].y, normlist[threadIdx.x + (NTHREADS * 4)].z,
    //    threadIdx.x + (NTHREADS * 5), normlist[threadIdx.x + (NTHREADS * 5)].x, normlist[threadIdx.x + (NTHREADS * 5)].y, normlist[threadIdx.x + (NTHREADS * 5)].z,
    //    threadIdx.x + (NTHREADS * 6), normlist[threadIdx.x + (NTHREADS * 6)].x, normlist[threadIdx.x + (NTHREADS * 6)].y, normlist[threadIdx.x + (NTHREADS * 6)].z,
    //    threadIdx.x + (NTHREADS * 7), normlist[threadIdx.x + (NTHREADS * 7)].x, normlist[threadIdx.x + (NTHREADS * 7)].y, normlist[threadIdx.x + (NTHREADS * 7)].z,
    //    threadIdx.x + (NTHREADS * 8), normlist[threadIdx.x + (NTHREADS * 8)].x, normlist[threadIdx.x + (NTHREADS * 8)].y, normlist[threadIdx.x + (NTHREADS * 8)].z,
    //    threadIdx.x + (NTHREADS * 9), normlist[threadIdx.x + (NTHREADS * 9)].x, normlist[threadIdx.x + (NTHREADS * 9)].y, normlist[threadIdx.x + (NTHREADS * 9)].z,
    //    threadIdx.x + (NTHREADS * 10), normlist[threadIdx.x + (NTHREADS * 10)].x, normlist[threadIdx.x + (NTHREADS * 10)].y, normlist[threadIdx.x + (NTHREADS * 10)].z,
    //    threadIdx.x + (NTHREADS * 11), normlist[threadIdx.x + (NTHREADS * 11)].x, normlist[threadIdx.x + (NTHREADS * 11)].y, normlist[threadIdx.x + (NTHREADS * 11)].z
    //    );
    __syncthreads();

#else
    float3 VerticesList[12];
    float3 NormalsList[12];

    CudaVertexLerp2(IsoValue, Value[0], Value[1], Field[0], Field[1], VerticesList[0], NormalsList[0]);
    CudaVertexLerp2(IsoValue, Value[1], Value[2], Field[1], Field[2], VerticesList[1], NormalsList[1]);
    CudaVertexLerp2(IsoValue, Value[2], Value[3], Field[2], Field[3], VerticesList[2], NormalsList[2]);
    CudaVertexLerp2(IsoValue, Value[3], Value[0], Field[3], Field[0], VerticesList[3], NormalsList[3]);

    CudaVertexLerp2(IsoValue, Value[4], Value[5], Field[4], Field[5], VerticesList[4], NormalsList[4]);
    CudaVertexLerp2(IsoValue, Value[5], Value[6], Field[5], Field[6], VerticesList[5], NormalsList[5]);
    CudaVertexLerp2(IsoValue, Value[6], Value[7], Field[6], Field[7], VerticesList[6], NormalsList[6]);
    CudaVertexLerp2(IsoValue, Value[7], Value[4], Field[7], Field[4], VerticesList[7], NormalsList[7]);

    CudaVertexLerp2(IsoValue, Value[0], Value[4], Field[0], Field[4], VerticesList[8], NormalsList[8]);
    CudaVertexLerp2(IsoValue, Value[1], Value[5], Field[1], Field[5], VerticesList[9], NormalsList[9]);
    CudaVertexLerp2(IsoValue, Value[2], Value[6], Field[2], Field[6], VerticesList[10], NormalsList[10]);
    CudaVertexLerp2(IsoValue, Value[3], Value[7], Field[3], Field[7], VerticesList[11], NormalsList[11]);
#endif

    // output triangle vertices
    uint NumVertices = tex1Dfetch<uint>(NumVerticesTex, CubeIndex);

    for (int i = 0; i < NumVertices; i++)
    {
        uint Edge = tex1Dfetch<uint>(TriTex, CubeIndex * 16 + i);

        uint Index = NumScannedVertices[Voxel] + i;

        if (Index < NumMaxVertices)
        {
#if USE_SHARED
            OutPositions[Index] = make_float4(VerticesList[(Edge * NTHREADS) + threadIdx.x], 1.0f);
            OutNormals[Index] = make_float4(NormalsList[(Edge * NTHREADS) + threadIdx.x], 0.0f);
            //printf("\Alpha[%u] index: %u, edge: %u, pos=(%f, %f, %f, %f), norm=(%f, %f, %f, %f)\OutNormalVector", Index, index, edge,
            //    pos[index].x, pos[index].y, pos[index].z, pos[index].w,
            //    norm[index].x, norm[index].y, norm[index].z, norm[index].w);
#else
            OutPositions[OutputIdx] = make_float4(VerticesList[Edge], 1.0f);
            OutNormals[OutputIdx] = make_float4(NormalsList[Edge], 0.0f);
#endif
        }
    }
}

// calculate triangle normal
__device__
float3 CudaCalculateNormal(float3* Vertex0, float3* Vertex1, float3* Vertex2)
{
    float3 Edge0 = *Vertex1 - *Vertex0;
    float3 Edge1 = *Vertex2 - *Vertex0;
    // note - it's faster to perform normalization in Vertex shader rather than here
    return cross(Edge0, Edge1);
}

// version that calculates flat surface normal for each triangle
__global__
void CudaGenerateTriangles2(float4* OutPositions,
    float4* OutNormals,
    uint* CompactedVoxelArray,
    uint* NumScannedVertices,
    uchar* Volumes,
    uint3 GridSize,
    uint3 GridSizeShift,
    uint3 GridSizeMask,
    float3 VoxelSize,
    float IsoValue,
    uint NumActiveVoxels,
    uint NumMaxVertices,
    hipTextureObject_t TriTex,
    hipTextureObject_t NumVerticesTex,
    hipTextureObject_t VolumeTex,
    float4* SortedPositions,
    uint* GridParticleIndices,
    uint* CellStarts,
    uint* CellEnds,
    uint NumFluidParticles,
    uint NumRenderingFluidParticles)
{
    uint BlockIdx = (blockIdx.y * gridDim.x) + blockIdx.x;
    uint Index = (BlockIdx * blockDim.x) + threadIdx.x;

    if (Index > NumActiveVoxels - 1)
    {
        Index = NumActiveVoxels - 1;
    }

#if SKIP_EMPTY_VOXELS
    uint Voxel = CompactedVoxelArray[Index];
#else
    uint Voxel = OutputIdx;
#endif

    // compute position in 3d grid
    uint3 GridPosition = CudaCalculateGridPosition(Voxel, GridSizeShift, GridSizeMask);

    float3 Position;
    Position.x = -1.0f + (GridPosition.x * VoxelSize.x);
    Position.y = -1.0f + (GridPosition.y * VoxelSize.y);
    Position.z = -1.0f + (GridPosition.z * VoxelSize.z);

    // calculate cell Vertex positions
    float3 Vertices[8];
    Vertices[0] = Position;
    Vertices[1] = Position + make_float3(VoxelSize.x, 0, 0);
    Vertices[2] = Position + make_float3(VoxelSize.x, VoxelSize.y, 0);
    Vertices[3] = Position + make_float3(0, VoxelSize.y, 0);
    Vertices[4] = Position + make_float3(0, 0, VoxelSize.z);
    Vertices[5] = Position + make_float3(VoxelSize.x, 0, VoxelSize.z);
    Vertices[6] = Position + make_float3(VoxelSize.x, VoxelSize.y, VoxelSize.z);
    Vertices[7] = Position + make_float3(0, VoxelSize.y, VoxelSize.z);


#if SAMPLE_VOLUME
    float Field[8];
    Field[0] = CudaSampleVolume(VolumeTex, Volumes, GridPosition, GridSize);
    Field[1] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(1, 0, 0), GridSize);
    Field[2] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(1, 1, 0), GridSize);
    Field[3] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(0, 1, 0), GridSize);
    Field[4] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(0, 0, 1), GridSize);
    Field[5] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(1, 0, 1), GridSize);
    Field[6] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(1, 1, 1), GridSize);
    Field[7] = CudaSampleVolume(VolumeTex, Volumes, GridPosition + make_uint3(0, 1, 1), GridSize);
#else
    // evaluate Field values
    float Field[8] = { FLT_MAX,
                        FLT_MAX,
                        FLT_MAX,
                        FLT_MAX,

                        FLT_MAX,
                        FLT_MAX,
                        FLT_MAX,
                        FLT_MAX, };

    for (int z = -1; z <= 1; ++z)
    {
        for (int y = -1; y <= 1; ++y)
        {
            for (int x = -1; x <= 1; ++x)
            {
                int3 NeighborPosition = make_int3(GridPosition) + make_int3(x, y, z);

                CudaClassifyVoxelsByGrid(NeighborPosition,
                    Field,
                    Position,
                    SortedPositions,
                    VoxelSize,
                    GridParticleIndices,
                    CellStarts,
                    CellEnds,
                    NumFluidParticles,
                    NumRenderingFluidParticles);
            }
        }
    }
#endif

    // recalculate flag
    uint CudeIndex;
    CudeIndex = uint(Field[0] < IsoValue);
    CudeIndex += uint(Field[1] < IsoValue) * 2u;
    CudeIndex += uint(Field[2] < IsoValue) * 4u;
    CudeIndex += uint(Field[3] < IsoValue) * 8u;
    CudeIndex += uint(Field[4] < IsoValue) * 16u;
    CudeIndex += uint(Field[5] < IsoValue) * 32u;
    CudeIndex += uint(Field[6] < IsoValue) * 64u;
    CudeIndex += uint(Field[7] < IsoValue) * 128u;

    // find the vertices where the surface intersects the cube

#if USE_SHARED
    // use shared memory to avoid using local
    __shared__ float3 VerticesList[12 * NTHREADS];

    VerticesList[threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[0], Vertices[1], Field[0], Field[1]);
    VerticesList[NTHREADS + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[1], Vertices[2], Field[1], Field[2]);
    VerticesList[(NTHREADS * 2) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[2], Vertices[3], Field[2], Field[3]);
    VerticesList[(NTHREADS * 3) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[3], Vertices[0], Field[3], Field[0]);
    VerticesList[(NTHREADS * 4) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[4], Vertices[5], Field[4], Field[5]);
    VerticesList[(NTHREADS * 5) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[5], Vertices[6], Field[5], Field[6]);
    VerticesList[(NTHREADS * 6) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[6], Vertices[7], Field[6], Field[7]);
    VerticesList[(NTHREADS * 7) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[7], Vertices[4], Field[7], Field[4]);
    VerticesList[(NTHREADS * 8) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[0], Vertices[4], Field[0], Field[4]);
    VerticesList[(NTHREADS * 9) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[1], Vertices[5], Field[1], Field[5]);
    VerticesList[(NTHREADS * 10) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[2], Vertices[6], Field[2], Field[6]);
    VerticesList[(NTHREADS * 11) + threadIdx.x] = CudaVertexLerp(IsoValue, Vertices[3], Vertices[7], Field[3], Field[7]);
    __syncthreads();
#else

    float3 VerticesList[12];

    VerticesList[0] = CudaVertexLerp(IsoValue, Value[0], Value[1], Field[0], Field[1]);
    VerticesList[1] = CudaVertexLerp(IsoValue, Value[1], Value[2], Field[1], Field[2]);
    VerticesList[2] = CudaVertexLerp(IsoValue, Value[2], Value[3], Field[2], Field[3]);
    VerticesList[3] = CudaVertexLerp(IsoValue, Value[3], Value[0], Field[3], Field[0]);

    VerticesList[4] = CudaVertexLerp(IsoValue, Value[4], Value[5], Field[4], Field[5]);
    VerticesList[5] = CudaVertexLerp(IsoValue, Value[5], Value[6], Field[5], Field[6]);
    VerticesList[6] = CudaVertexLerp(IsoValue, Value[6], Value[7], Field[6], Field[7]);
    VerticesList[7] = CudaVertexLerp(IsoValue, Value[7], Value[4], Field[7], Field[4]);

    VerticesList[8] = CudaVertexLerp(IsoValue, Value[0], Value[4], Field[0], Field[4]);
    VerticesList[9] = CudaVertexLerp(IsoValue, Value[1], Value[5], Field[1], Field[5]);
    VerticesList[10] = CudaVertexLerp(IsoValue, Value[2], Value[6], Field[2], Field[6]);
    VerticesList[11] = CudaVertexLerp(IsoValue, Value[3], Value[7], Field[3], Field[7]);
#endif

    // output triangle vertices
    uint NumVertices = tex1Dfetch<uint>(NumVerticesTex, CudeIndex);

    for (int VertexIdx = 0; VertexIdx < NumVertices; VertexIdx += 3)
    {
        uint OutputIdx = NumScannedVertices[Voxel] + VertexIdx;

        float3* OutputVertices[3];
        uint Edge;
        Edge = tex1Dfetch<uint>(TriTex, (CudeIndex * 16) + VertexIdx);
#if USE_SHARED
        OutputVertices[0] = &VerticesList[(Edge * NTHREADS) + threadIdx.x];
#else
        Value[0] = &VerticesList[Edge];
#endif

        Edge = tex1Dfetch<uint>(TriTex, (CudeIndex * 16) + VertexIdx + 1);
#if USE_SHARED
        OutputVertices[1] = &VerticesList[(Edge * NTHREADS) + threadIdx.x];
#else
        Value[1] = &VerticesList[Edge];
#endif

        Edge = tex1Dfetch<uint>(TriTex, (CudeIndex * 16) + VertexIdx + 2);
#if USE_SHARED
        OutputVertices[2] = &VerticesList[(Edge * NTHREADS) + threadIdx.x];
#else
        Value[2] = &VerticesList[Edge];
#endif

        // calculate triangle surface normal
        float3 Normal = CudaCalculateNormal(OutputVertices[0], OutputVertices[1], OutputVertices[2]);

        if (OutputIdx < (NumMaxVertices - 3))
        {
            OutPositions[OutputIdx] = make_float4(*OutputVertices[0], 1.0f);
            OutNormals[OutputIdx] = make_float4(Normal, 0.0f);

            OutPositions[OutputIdx + 1] = make_float4(*OutputVertices[1], 1.0f);
            OutNormals[OutputIdx + 1] = make_float4(Normal, 0.0f);

            OutPositions[OutputIdx + 2] = make_float4(*OutputVertices[2], 1.0f);
            OutNormals[OutputIdx + 2] = make_float4(Normal, 0.0f);
        }
    }
}

__device__
float CudaMcComputeDensities(int3 GridPosition,
    float3 Position,
    float4* SortedPositions,
    uint NumFluidParticles,
    uint* CellStarts,
    uint* CellEnds)
{
    uint GridHash = CudaMcCalculateGridHash(GridPosition);

    uint StartIndex = CellStarts[GridHash];

    float Density = 0.0f;
    if (StartIndex != 0xffffffff)
    {
        uint EndIndex = CellEnds[GridHash];

        for (uint NeighborIdx = StartIndex; NeighborIdx < EndIndex; ++NeighborIdx)
        {
            float3 Rij = Position - make_float3(SortedPositions[NeighborIdx]);
            float R2 = lengthSquared(Rij);

            if (R2 < gParameters.SupportRadiusSquared)
            {
                if (NeighborIdx < NumFluidParticles)
                {
                    Density += gParameters.ParticleMass * CudaKernelPoly6ByDistanceSquared(R2);
                }
                else
                {
                    Density += gParameters.BoundaryParticleMass * CudaKernelPoly6ByDistanceSquared(R2);
                }
            }
        }
    }
    return Density;
}

__global__
void CudaCreateVolumeFromMassAndDensitiesDevice(uchar* OutVolumes,
    uint3 GridSize,
    uint3 GridSizeShift,
    uint3 GridSizeMask,
    float3 VoxelSize,
    uint NumFluidParticles,
    float4* SortedPositions,
    uint* GridParticleIndices,
    uint* CellStarts,
    uint* CellEnds)
{
    uint BlockIdx = (blockIdx.y * gridDim.x) + blockIdx.x;
    uint Index = (BlockIdx * blockDim.x) + threadIdx.x;

    uint3 GridPosition = CudaCalculateGridPosition(Index, GridSizeShift, GridSizeMask);

    float3 Position;
    Position.x = -1.0f + (GridPosition.x * VoxelSize.x);
    Position.y = -1.0f + (GridPosition.y * VoxelSize.y);
    Position.z = -1.0f + (GridPosition.z * VoxelSize.z);

    float Density = 0.0f;
    for (int z = -1; z <= 1; ++z)
    {
        for (int y = -1; y <= 1; ++y)
        {
            for (int x = -1; x <= 1; ++x)
            {
                int3 NeighborPosition = make_int3(GridPosition) + make_int3(x, y, z);

                Density += CudaMcComputeDensities(NeighborPosition,
                    Position,
                    SortedPositions,
                    NumFluidParticles,
                    CellStarts,
                    CellEnds);
            }
        }
    }

    OutVolumes[Index] = gParameters.ParticleMass / Density;
}
#pragma endregion

int main()
{
    return 0;
}


#pragma region SphExtern
//particleSystem_cuda.cu
extern "C"
{

    void CudaInit(int Argc, char** Argv)
    {
        int devID;

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        devID = findCudaDevice(Argc, (const char**)Argv);

        if (devID < 0)
        {
            printf("No CUDA Capable devices found, exiting...\n");
            exit(EXIT_SUCCESS);
        }
    }

    void CudaAllocateArray(void** DevicePtr, size_t Size)
    {
        checkCudaErrors(hipMalloc(DevicePtr, Size));
    }

    void CudaFreeArray(void* DevicePtr)
    {
        checkCudaErrors(hipFree(DevicePtr));
    }

    void CudaThreadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    void CudaCopyArrayToDevice(void* DeviceArray, const void* HostArray, size_t Offset, size_t Size)
    {
        checkCudaErrors(hipMemcpy((char*)DeviceArray + Offset, HostArray, Size, hipMemcpyHostToDevice));
    }

    void CudaSetParameters(CudaSimParams* HostParameters)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gParameters), HostParameters, sizeof(CudaSimParams)));
    }

    //Round a / b to nearest higher integer value
    uint CudaDivideUp(uint A, uint B)
    {
        return (A % B != 0) ? (A / B + 1) : (A / B);
    }

    // compute grid and thread block size for a given number of elements
    void CudaComputeGridSize(uint Size, uint BlockSize, uint& OutNumBlocks, uint& OutNumThreads)
    {
        OutNumThreads = min(BlockSize, Size);
        OutNumBlocks = CudaDivideUp(Size, OutNumThreads);
    }



    void CudaIntegrateSystem(float* Positions, float* Velocities, uint NumParticles)
    {
        thrust::device_ptr<float4> DevicePositions((float4*)Positions);
        thrust::device_ptr<float4> DeviceVelocities((float4*)Velocities);

        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(DevicePositions, DeviceVelocities)),
            thrust::make_zip_iterator(thrust::make_tuple(DevicePositions + NumParticles, DeviceVelocities + NumParticles)),
            IntegrateFunctor());
    }

    void CudaCalculateHashes(uint* OutGridParticleHashes,
        uint* OutGridParticleIndices,
        float* Positions,
        uint   NumParticles)
    {
        uint NumThreads;
        uint NumBlocks;
        CudaComputeGridSize(NumParticles, 256u, NumBlocks, NumThreads);

        // execute the kernel
        CudaCalculateHashDevice << <NumBlocks, NumThreads >> > (OutGridParticleHashes,
            OutGridParticleIndices,
            (float4*)Positions,
            NumParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void CudaMcCalculateHashes(uint* OutGridParticleHashes,
        uint* OutGridParticleIndices,
        float* Positions,
        uint   NumParticles)
    {
        uint NumThreads;
        uint NumBlocks;
        CudaComputeGridSize(NumParticles, 256u, NumBlocks, NumThreads);

        // execute the kernel
        CudaMcCalculateHashDevice << <NumBlocks, NumThreads >> > (OutGridParticleHashes,
            OutGridParticleIndices,
            (float4*)Positions,
            NumParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void CudaReorderDataAndFindCellStart(uint* OutCellStarts,
        uint* OutCellEnds,
        float* OutSortedPositions,
        float* OutSortedVelocities,
        uint* GridParticleHashes,
        uint* GridParticleIndices,
        float* Positions,
        float* Velocities,
        uint   NumParticles,
        uint   NumCells)
    {
        uint NumThreads;
        uint NumBlocks;
        CudaComputeGridSize(NumParticles, 256u, NumBlocks, NumThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(OutCellStarts, 0xffffffff, NumCells * sizeof(uint)));

        uint SmemSize = sizeof(uint) * (NumThreads + 1);
        CudaReorderDataAndFindCellStartDevice << <NumBlocks, NumThreads, SmemSize >> > (OutCellStarts,
            OutCellEnds,
            (float4*)OutSortedPositions,
            (float4*)OutSortedVelocities,
            GridParticleHashes,
            GridParticleIndices,
            (float4*)Positions,
            (float4*)Velocities,
            NumParticles);
        getLastCudaError("Kernel execution failed: CudaReorderDataAndFindCellStartDevice");

    }

    void CudaMcReorderDataAndFindCellStart(uint* OutCellStarts,
        uint* OutCellEnds,
        float* OutSortedPositions,
        uint* GridParticleHashes,
        uint* GridParticleIndices,
        float* Positions,
        uint   NumParticles,
        uint   NumVoxels)
    {
        uint NumThreads;
        uint NumBlocks;
        CudaComputeGridSize(NumParticles, 256u, NumBlocks, NumThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(OutCellStarts, 0xffffffff, NumVoxels * sizeof(uint)));

        uint SmemSize = sizeof(uint) * (NumThreads + 1);
        CudaMcReorderDataAndFindCellStartDevice << <NumBlocks, NumThreads, SmemSize >> > (OutCellStarts,
            OutCellEnds,
            (float4*)OutSortedPositions,
            GridParticleHashes,
            GridParticleIndices,
            (float4*)Positions,
            NumParticles);
        getLastCudaError("Kernel execution failed: CudaReorderDataAndFindCellStartDevice");

    }

    void CudaComputeDensitiesAndPressures(float* OutDensities,
        float* OutPressures,
        float* SortedPositions,
        uint* GridParticleIndices,
        uint* CellStarts,
        uint* CellEnds,
        uint   NumBoundaryParticles,
        uint   NumFluidParticles,
        uint   NumRenderingFluidParticles,
        uint   NumParticles)
    {

        // thread per particle
        uint NumThreads;
        uint NumBlocks;
        CudaComputeGridSize(NumParticles, 64u, NumBlocks, NumThreads);

        // execute the kernel
        CudaComputeDensitiesAndPressuresDevice << <NumBlocks, NumThreads >> > (OutDensities,
            OutPressures,
            (float4*)SortedPositions,
            GridParticleIndices,
            CellStarts,
            CellEnds,
            NumBoundaryParticles,
            NumFluidParticles,
            NumRenderingFluidParticles,
            NumParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

    }

    void CudaComputeAllForcesAndVelocities(float* OutVelocities,
        float* OutForces,
        float* OutPressureForces,
        float* OutViscosityForces,
        float* SortedPositions, // input: sorted positions
        float* SortedVelocities,
        float* Densities,
        float* Pressures,
        uint* GridParticleIndices,    // input: sorted particle indices
        uint* CellStarts,
        uint* CellEnds,
        uint   NumFluidParticles,
        uint   NumRenderingFluidParticles,
        uint   NumParticles)
    {
        // thread per particle
        uint NumThreads;
        uint NumBlocks;
        CudaComputeGridSize(NumParticles, 64u, NumBlocks, NumThreads);

        // execute the kernel
        CudaComputeAllForcesAndVelocitiesDevice << <NumBlocks, NumThreads >> > ((float4*)OutVelocities,
            (float4*)OutForces,
            (float4*)OutPressureForces,
            (float4*)OutViscosityForces,
            (float4*)SortedPositions,   // input: sorted positions
            (float4*)SortedVelocities,
            Densities,
            Pressures,
            GridParticleIndices,         // input: sorted particle indices
            CellStarts,
            CellEnds,
            NumFluidParticles,
            NumRenderingFluidParticles,
            NumParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    void CudaComputeForcesAndVelocities(float* OutVelocities,
        float* OutForces,
        float* SortedPositions, // input: sorted positions
        float* SortedVelocities,
        float* Densities,
        float* Pressures,
        uint* GridParticleIndices,    // input: sorted particle indices
        uint* CellStarts,
        uint* CellEnds,
        uint   NumFluidParticles,
        uint   NumRenderingFluidParticles,
        uint   NumParticles)
    {
        // thread per particle
        uint NumThreads;
        uint NumBlocks;
        CudaComputeGridSize(NumParticles, 64u, NumBlocks, NumThreads);

        // execute the kernel
        CudaComputeForcesAndVelocitiesDevice << <NumBlocks, NumThreads >> > ((float4*)OutVelocities,
            (float4*)OutForces,
            (float4*)SortedPositions,   // input: sorted positions
            (float4*)SortedVelocities,
            Densities,
            Pressures,
            GridParticleIndices,         // input: sorted particle indices
            CellStarts,
            CellEnds,
            NumFluidParticles,
            NumRenderingFluidParticles,
            NumParticles);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }


    void CudaSortParticles(uint* DeviceGridParticleHashes, uint* DeviceGridParticleIndices, uint NumParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(DeviceGridParticleHashes),
            thrust::device_ptr<uint>(DeviceGridParticleHashes + NumParticles),
            thrust::device_ptr<uint>(DeviceGridParticleIndices));
    }
#pragma endregion

#pragma region MarchingCubesExtern
    // marching cubes
    void CudaAllocateTextures(uint** DeviceEdgeTable, uint** DeviceTriTable, uint** DeviceNumVerticesTable)
    {
        checkCudaErrors(hipMalloc((void**)DeviceEdgeTable, 256u * sizeof(uint)));
        checkCudaErrors(hipMemcpy((void*)*DeviceEdgeTable, (void*)gEdgesTable, 256 * sizeof(uint), hipMemcpyHostToDevice));
        hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

        checkCudaErrors(hipMalloc((void**)DeviceTriTable, 256u * 16u * sizeof(uint)));
        checkCudaErrors(hipMemcpy((void*)*DeviceTriTable, (void*)gTrianglesTable, 256u * 16u * sizeof(uint), hipMemcpyHostToDevice));

        hipResourceDesc            TextureResource;
        memset(&TextureResource, 0, sizeof(hipResourceDesc));

        TextureResource.resType = hipResourceTypeLinear;
        TextureResource.res.linear.devPtr = *DeviceTriTable;
        TextureResource.res.linear.sizeInBytes = 256u * 16u * sizeof(uint);
        TextureResource.res.linear.desc = ChannelDesc;

        hipTextureDesc             TexDesc;
        memset(&TexDesc, 0, sizeof(hipTextureDesc));

        TexDesc.normalizedCoords = false;
        TexDesc.filterMode = hipFilterModePoint;
        TexDesc.addressMode[0] = hipAddressModeClamp;
        TexDesc.readMode = hipReadModeElementType;

        checkCudaErrors(hipCreateTextureObject(&gTriTex, &TextureResource, &TexDesc, nullptr));

        checkCudaErrors(hipMalloc((void**)DeviceNumVerticesTable, 256u * sizeof(uint)));
        checkCudaErrors(hipMemcpy((void*)*DeviceNumVerticesTable, (void*)gNumVerticesTable, 256u * sizeof(uint), hipMemcpyHostToDevice));

        memset(&TextureResource, 0, sizeof(hipResourceDesc));

        TextureResource.resType = hipResourceTypeLinear;
        TextureResource.res.linear.devPtr = *DeviceNumVerticesTable;
        TextureResource.res.linear.sizeInBytes = 256 * sizeof(uint);
        TextureResource.res.linear.desc = ChannelDesc;

        memset(&TexDesc, 0, sizeof(hipTextureDesc));

        TexDesc.normalizedCoords = false;
        TexDesc.filterMode = hipFilterModePoint;
        TexDesc.addressMode[0] = hipAddressModeClamp;
        TexDesc.readMode = hipReadModeElementType;

        checkCudaErrors(hipCreateTextureObject(&gNumVerticesTex, &TextureResource, &TexDesc, nullptr));
    }

    void CudaCreateVolumeTexture(uchar* DeviceVolumes, size_t BufferSize)
    {
        hipResourceDesc            TextureResourceDesc;
        memset(&TextureResourceDesc, 0, sizeof(hipResourceDesc));

        TextureResourceDesc.resType = hipResourceTypeLinear;
        TextureResourceDesc.res.linear.devPtr = DeviceVolumes;
        TextureResourceDesc.res.linear.sizeInBytes = BufferSize;
        TextureResourceDesc.res.linear.desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

        hipTextureDesc             TextureDesc;
        memset(&TextureDesc, 0, sizeof(hipTextureDesc));

        TextureDesc.normalizedCoords = false;
        TextureDesc.filterMode = hipFilterModePoint;
        TextureDesc.addressMode[0] = hipAddressModeClamp;
        TextureDesc.readMode = hipReadModeNormalizedFloat;

        checkCudaErrors(hipCreateTextureObject(&gVolumeTex, &TextureResourceDesc, &TextureDesc, nullptr));
    }

    void CudaDestroyAllTextureObjects()
    {
        checkCudaErrors(hipDestroyTextureObject(gTriTex));
        checkCudaErrors(hipDestroyTextureObject(gNumVerticesTex));
        checkCudaErrors(hipDestroyTextureObject(gVolumeTex));
    }

    void CudaLaunchClassifyVoxels(dim3 Grid,
        dim3 Threads,
        uint* OutVoxelVertices,
        uint* OutOccupiedVoxels,
        uchar* Volumes,
        uint3 GridSize,
        uint3 GridSizeShift,
        uint3 GridSizeMask,
        uint NumVoxels,
        float3 VoxelSize,
        float IsoValue,
        float* SortedPositions,
        uint* GridParticleIndices,
        uint* CellStarts,
        uint* CellEnds,
        uint NumFluidParticles,
        uint NumRenderingFluidParticles)
    {
        // calculate number of vertices need per voxel
        CudaClassifyVoxels << <Grid, Threads >> > (OutVoxelVertices,
            OutOccupiedVoxels,
            Volumes,
            GridSize,
            GridSizeShift,
            GridSizeMask,
            NumVoxels,
            VoxelSize,
            IsoValue,
            gNumVerticesTex,
            gVolumeTex,
            reinterpret_cast<float4*>(SortedPositions),
            GridParticleIndices,
            CellStarts,
            CellEnds,
            NumFluidParticles,
            NumRenderingFluidParticles);
        getLastCudaError("CudaClassifyVoxels failed");
    }

    void CudaLaunchCompactVoxels(dim3 Grid, dim3 Threads, uint* OutCompactedVoxelArray, uint* OccupiedVoxels, uint* OccupiedScanVoxels, uint NumVoxels)
    {
        CudaCompactVoxels << <Grid, Threads >> > (OutCompactedVoxelArray,
            OccupiedVoxels,
            OccupiedScanVoxels,
            NumVoxels);
        getLastCudaError("CudaCompactVoxels failed");
    }

    void CudaLaunchGenerateTriangles(dim3 Grid,
        dim3 Threads,
        float4* OutPositions,
        float4* OutNormals,
        uint* CompactedVoxelArray,
        uint* NumScannedVertices,
        uint3 GridSize,
        uint3 GridSizeShift,
        uint3 GridSizeMask,
        float3 VoxelSize,
        float IsoValue,
        uint NumActiveVoxels,
        uint NumMaxVertices,
        float* SortedPositions,
        uint* GridParticleIndices,
        uint* CellStarts,
        uint* CellEnds,
        uint NumFluidParticles,
        uint NumRenderingFluidParticles)
    {
        CudaGenerateTriangles << <Grid, Threads >> > (OutPositions,
            OutNormals,
            CompactedVoxelArray,
            NumScannedVertices,
            GridSize,
            GridSizeShift,
            GridSizeMask,
            VoxelSize,
            IsoValue,
            NumActiveVoxels,
            NumMaxVertices,
            gTriTex,
            gNumVerticesTex,
            reinterpret_cast<float4*>(SortedPositions),
            GridParticleIndices,
            CellStarts,
            CellEnds,
            NumFluidParticles,
            NumRenderingFluidParticles);
        getLastCudaError("CudaGenerateTriangles failed");
    }

    void CudaLaunchGenerateTriangles2(dim3 Grid,
        dim3 Threads,
        float4* OutPositions,
        float4* OutNormals,
        uint* CompactedVoxelArray,
        uint* NumScannedVertices,
        uchar* Volumes,
        uint3 GridSize,
        uint3 GridSizeShift,
        uint3 GridSizeMask,
        float3 VoxelSize,
        float IsoValue,
        uint NumActiveVoxels,
        uint NumMaxVertices,
        float4* SortedPositions,
        uint* GridParticleIndices,
        uint* CellStarts,
        uint* CellEnds,
        uint NumFluidParticles,
        uint NumRenderingFluidParticles)
    {
        CudaGenerateTriangles2 << <Grid, Threads >> > (OutPositions,
            OutNormals,
            CompactedVoxelArray,
            NumScannedVertices,
            Volumes,
            GridSize,
            GridSizeShift,
            GridSizeMask,
            VoxelSize,
            IsoValue,
            NumActiveVoxels,
            NumMaxVertices,
            gTriTex,
            gNumVerticesTex,
            gVolumeTex,
            SortedPositions,
            GridParticleIndices,
            CellStarts,
            CellEnds,
            NumFluidParticles,
            NumRenderingFluidParticles);
        getLastCudaError("CudaGenerateTriangles2 failed");
    }

    void CudaThrustScanWrapper(unsigned int* Outputs, unsigned int* Inputs, unsigned int NumElements)
    {
        thrust::exclusive_scan(thrust::device_ptr<unsigned int>(Inputs),
            thrust::device_ptr<unsigned int>(Inputs + NumElements),
            thrust::device_ptr<unsigned int>(Outputs));
    }

    void CudaCreateVolumeFromMassAndDensities(dim3 Grid,
        dim3 Threads,
        uchar* OutVolumes,
        uint3 GridSize,
        uint3 GridSizeShift,
        uint3 GridSizeMask,
        float3 VoxelSize,
        uint NumFluidParticles,
        float4* SortedPositions,
        uint* GridParticleIndices,
        uint* CellStarts,
        uint* CellEnds)
    {
        CudaCreateVolumeFromMassAndDensitiesDevice << <Grid, Threads >> > (OutVolumes,
            GridSize,
            GridSizeShift,
            GridSizeMask,
            VoxelSize,
            NumFluidParticles,
            SortedPositions,
            GridParticleIndices,
            CellStarts,
            CellEnds);
    }
}   // extern "C"
#pragma endregion